#include <stdio.h>
#include <array>
#include <vector>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{
  printf("Using 3 GPU for test\n"); fflush(stdout);
  // managing 4 devices
  int size = 32*1024*1024;
  const std::vector<int> devs = { 0,1,2,3,4,5,6,7 };
  ncclComm_t comms[devs.size()];

  printf("Allocate send & recv buffer\n"); fflush(stdout);
  // allocating and initializing device buffers
  float** sendbuff = (float**)malloc(devs.size() * sizeof(float*));
  float** recvbuff = (float**)malloc(devs.size() * sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*(devs.size()));

  // Allocate memory
  for ( int i = 0; i < devs.size(); ++i ) 
  {
    CUDACHECK(hipSetDevice( devs[i] ));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }

  //initializing NCCL
  printf("Initial comm\n"); fflush(stdout);
  NCCLCHECK(ncclCommInitAll(comms, devs.size(), devs.data()));

   //calling NCCL communication API. Group API is required when using
   //multiple devices per thread
  printf("Run allreduce\n"); fflush(stdout);
  NCCLCHECK(ncclGroupStart());
  for ( int i = 0; i < devs.size(); ++i ) 
  {
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, \
      ncclSum, comms[i], s[i]));
  }
  NCCLCHECK(ncclGroupEnd());


  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for ( int i = 0; i < devs.size(); ++i ) 
  {
    CUDACHECK(hipSetDevice(devs[i]));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  //free device buffers
  for ( int i = 0; i < devs.size(); ++i )
  {
    CUDACHECK(hipSetDevice(devs[i]));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }


  //finalizing NCCL
  for ( int i = 0; i < devs.size(); ++i )
  {
      ncclCommDestroy(comms[i]);
  }

  printf("Success \n");
  return 0;
}